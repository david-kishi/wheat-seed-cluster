/******************************************************************************
* FILE: wheat_cluster.cu
* DESCRIPTION:
*   A simple cuda program to compute k-means cluster of a variety of wheat
*   wheat seeds.
* AUTHOR: David Nguyen
* CONTACT: david@knytes.com 
* REVISED: 14/05/2020
******************************************************************************/
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define AMT_OF_CLUSTERS = 3

//TODO Device function for k-means clustering
// Should return the new points of the centroids

int main(){
  // Read in dataset and store into array.
  // 210 total - 70 per class - put 10 random from each class
  float     *area,
            *perimeter,
            *compactness,
            *lenKernel,
            *widKernel,
            *asymCoef,
            *lenKernelGroove,
            *cat,
            *t_area,
            *t_perimeter,
            *t_compactness,
            *t_lenKernel,
            *t_widKernel,
            *t_asymCoef,
            *t_lenKernelGroove,
            *t_cat;

  // Allocate memory space to variables
  area = (float*)malloc(180*sizeof(float));
  perimeter = (float*)malloc(180*sizeof(float));
  compactness = (float*)malloc(180*sizeof(float));
  lenKernel = (float*)malloc(180*sizeof(float));
  widKernel = (float*)malloc(180*sizeof(float));
  asymCoef = (float*)malloc(180*sizeof(float));
  lenKernelGroove = (float*)malloc(180*sizeof(float));
  cat = (float*)malloc(180*sizeof(float));
  t_area = (float*)malloc(30*sizeof(float));
  t_perimeter = (float*)malloc(30*sizeof(float));
  t_compactness = (float*)malloc(30*sizeof(float));
  t_lenKernel = (float*)malloc(30*sizeof(float));
  t_widKernel = (float*)malloc(30*sizeof(float));
  t_asymCoef = (float*)malloc(30*sizeof(float));
  t_lenKernelGroove = (float*)malloc(30*sizeof(float));
  t_cat = (float*)malloc(30*sizeof(float));

  /**
  * Read in data from text file and store into training and testing sets
  */
  FILE  *fp; // File object
  float fTmp; // temporarily store float
  int   typeCount = 0, // keep count of feature
        trainingCount = 0, // keep count of size of training set
        entries = 0; // keep count of entry
  const char* filename = "seeds_dataset.txt"; // file name

  // Open file
  fp = fopen(filename, "r");

  // Check if file exists
  if (fp == NULL){
    printf("Could not open file %s",filename);
    return 1;
  }

  printf("READING DATASET\n%d: ", entries+1);
  while(entries != 210){
    fscanf(fp,"%6f",&fTmp);
    printf("%.4f ",fTmp);

    // Store value into proper array
    switch(typeCount){
      case 0: // area
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_area[trainingCount] = fTmp;
        }else{
          area[entries-trainingCount] = fTmp;
        }
        typeCount++;
      break;

      case 1: // perimeter
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_perimeter[trainingCount] = fTmp;
        }else{
          perimeter[entries-trainingCount]=fTmp;
        }
        typeCount++;
      break;

      case 2: // compactness
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_compactness[trainingCount] = fTmp;
        }else{
          compactness[entries-trainingCount]=fTmp;
        }
        typeCount++;
      break;

      case 3: // length of kernel
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_lenKernel[trainingCount] = fTmp;
        }else{
          lenKernel[entries-trainingCount]=fTmp;
        }
        typeCount++;
      break;

      case 4: // width of kernel
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_widKernel[trainingCount] = fTmp;
        }else{
          widKernel[entries-trainingCount]=fTmp;
        }
        typeCount++;
      break;

      case 5: // asymmetry coefficient
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_asymCoef[trainingCount] = fTmp;
        }else{
          asymCoef[entries-trainingCount]=fTmp;
        }
        typeCount++;
      break;

      case 6: // length of kernel groove
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_lenKernelGroove[trainingCount] = fTmp;          
        }else{
          lenKernelGroove[entries-trainingCount]=fTmp;
        };
        typeCount++;
      break;

      case 7: // class
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_cat[trainingCount] = fTmp;
          trainingCount++;
        }else{
          cat[entries-trainingCount]=fTmp;
        }
        typeCount = 0;
        entries++;
        if(entries != 210){
          printf("\n%d: ", entries+1);
        }else{
          printf("\n");
        }
      break;
      
      default:
        printf("Invalid entry during data read in.\n");
    }
  }

  // Close file
  fclose(fp);

  //TODO Call device function and run for declared amount of epochs

  //TODO Run new centroids against testing set and return accuracy
  

  // Free memory
  free(area);
  free(perimeter);
  free(compactness);
  free(lenKernel);
  free(widKernel);
  free(asymCoef);
  free(lenKernelGroove);
  free(cat);
  free(t_area);
  free(t_perimeter);
  free(t_compactness);
  free(t_lenKernel);
  free(t_widKernel);
  free(t_asymCoef);
  free(t_lenKernelGroove);
  free(t_cat);

  return 0;
}