/******************************************************************************
* FILE: wheat_cluster.cu
* DESCRIPTION:
*   A simple cuda program to compute k-means cluster of a variety of wheat
*   wheat seeds.
* AUTHOR: David Nguyen
* CONTACT: david@knytes.com 
* REVISED: 14/05/2020
******************************************************************************/
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define AMT_OF_CLUSTERS = 3

/**
 * Device function to perform 1 pass of k-means clustering
 *  @param  {float*} x - array of a feature
 *  @param  {float*} y - array of a feature
 *  @param  {int} amt - amount of values (30 or 180)
 *  @param  {float*} cents - centroid coordinates
 *  @param  {float*} predict - array to hold predicted class values
 */
__global__
void kMeansClustering(float *x, float *y, int amt, float *cents, float *predict){
  for(int i = 0; i < amt; i++){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute Euclidean Distances
    float eucD_a = sqrt(pow(x[tid] - cents[0], 2) + pow(y[tid] - cents[1], 2));
    float eucD_b = sqrt(pow(x[tid] - cents[2], 2) + pow(y[tid] - cents[3], 2));
    float eucD_c = sqrt(pow(x[tid] - cents[4], 2) + pow(y[tid] - cents[5], 2));

    // Compare Euclidean Distances
    if(eucD_a <= eucD_b && eucD_a <= eucD_c){
      predict[tid] = 1;
    }else if(eucD_b <= eucD_a && eucD_b <= eucD_c){
      predict[tid] = 2;
    }else{
      predict[tid] = 3;
    }
  }
}

/**
 * Host function to check predictions
 *  @param  {float*} pred - array of predictions
 *  @param  {float*} act - array of actual class
 *  @param  {int} amt - amount of predicted values (30 or 180)
 */
__host__
void checkPredictions(float *pred, float *act, int amt){
  int falseCnt = 0;
  for(int i = 0; i < amt; i++){
    if(pred[i] != act[i]) { falseCnt++; }
  }
  printf("Correct Predictions: %d/%d\n",amt-falseCnt, amt);
  printf("Accuracy: %.4f\n", 1.0*(amt-falseCnt)/amt);
}

/**
 * Host function to update centroid coordinates
 *  @param  {float*} x - array of a feature
 *  @param  {float*} y - array of a feature
 *  @param  {int} amt - amount of predicted values (30 or 180)
 */
__host__
void updateCentroids(float *x, float *y, float *pred, float *cent){
  float centA_x = 0,
        centA_y = 0,
        centB_x = 0,
        centB_y = 0,
        centC_x = 0,
        centC_y = 0;
  int oneCnt = 0,
      twoCnt = 0,
      threeCnt = 0;

  
  // Sum of values
  for(int i = 0; i < 180; i++){
    if(pred[i] == 1.0){
      centA_x += x[i];
      centA_y += y[i];
      oneCnt++;
    }else if(pred[i] == 2.0){
      centB_x += x[i];
      centB_y += y[i];
      twoCnt++;
    }else{
      centC_x += x[i];
      centC_y += y[i];
      threeCnt++;
    }
  }

  // Compute new centroids
  cent[0] = centA_x / oneCnt;
  cent[1] = centA_y / oneCnt;
  cent[2] = centB_x / twoCnt;
  cent[3] = centB_y / twoCnt;
  cent[4] = centC_x / threeCnt;
  cent[5] = centC_y / threeCnt;
}

int main(){
  // Read in dataset and store into array.
  // 210 total - 70 per class - put 10 random from each class
  float *area,
        *perimeter,
        *compactness,
        *lenKernel,
        *widKernel,
        *asymCoef,
        *lenKernelGroove,
        *cat,
        *t_area,
        *t_perimeter,
        *t_compactness,
        *t_lenKernel,
        *t_widKernel,
        *t_asymCoef,
        *t_lenKernelGroove,
        *t_cat,
        *predicted_180,
        *predicted_30,
        *predicted_180_d,
        *predicted_30_d,
        *centroids,
        *centroids_d,
        *a,
        *b;

  // Allocate memory space to variables
  area = (float*)malloc(180*sizeof(float));
  perimeter = (float*)malloc(180*sizeof(float));
  compactness = (float*)malloc(180*sizeof(float));
  lenKernel = (float*)malloc(180*sizeof(float));
  widKernel = (float*)malloc(180*sizeof(float));
  asymCoef = (float*)malloc(180*sizeof(float));
  lenKernelGroove = (float*)malloc(180*sizeof(float));
  cat = (float*)malloc(180*sizeof(float));
  t_area = (float*)malloc(30*sizeof(float));
  t_perimeter = (float*)malloc(30*sizeof(float));
  t_compactness = (float*)malloc(30*sizeof(float));
  t_lenKernel = (float*)malloc(30*sizeof(float));
  t_widKernel = (float*)malloc(30*sizeof(float));
  t_asymCoef = (float*)malloc(30*sizeof(float));
  t_lenKernelGroove = (float*)malloc(30*sizeof(float));
  t_cat = (float*)malloc(30*sizeof(float));
  predicted_180 = (float*)malloc(180*sizeof(float));
  predicted_30 = (float*)malloc(30*sizeof(float));
  centroids = (float*)malloc(6*sizeof(float));

  /**
  * Read in data from text file and store into training and testing sets
  */
  FILE  *fp; // File object
  float fTmp; // temporarily store float
  int   typeCount = 0, // keep count of feature
        trainingCount = 0, // keep count of size of training set
        entries = 0; // keep count of entry

  // Open file
  fp = fopen("data/seeds_dataset.txt", "r");

  // Check if file exists
  if (fp == NULL){
    printf("Could not open file %s","seeds_dataset.txt");
    return 1;
  }

  printf("Reading in dataset...\n");
  while(entries != 210){
    fscanf(fp,"%6f",&fTmp);

    // Store value into proper array
    switch(typeCount){
      case 0: // area
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_area[trainingCount] = fTmp;
        }else{
          area[entries-trainingCount] = fTmp;
        }
        typeCount++;
      break;

      case 1: // perimeter
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_perimeter[trainingCount] = fTmp;
        }else{
          perimeter[entries-trainingCount]=fTmp;
        }
        typeCount++;
      break;

      case 2: // compactness
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_compactness[trainingCount] = fTmp;
        }else{
          compactness[entries-trainingCount]=fTmp;
        }
        typeCount++;
      break;

      case 3: // length of kernel
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_lenKernel[trainingCount] = fTmp;
        }else{
          lenKernel[entries-trainingCount]=fTmp;
        }
        typeCount++;
      break;

      case 4: // width of kernel
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_widKernel[trainingCount] = fTmp;
        }else{
          widKernel[entries-trainingCount]=fTmp;
        }
        typeCount++;
      break;

      case 5: // asymmetry coefficient
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_asymCoef[trainingCount] = fTmp;
        }else{
          asymCoef[entries-trainingCount]=fTmp;
        }
        typeCount++;
      break;

      case 6: // length of kernel groove
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_lenKernelGroove[trainingCount] = fTmp;          
        }else{
          lenKernelGroove[entries-trainingCount]=fTmp;
        };
        typeCount++;
      break;

      case 7: // class
        if((entries >= 60 && entries < 70) ||
            (entries >= 130 && entries < 140) ||
            (entries >= 200 && entries < 210)){
          t_cat[trainingCount] = fTmp;
          trainingCount++;
        }else{
          cat[entries-trainingCount]=fTmp;
        }
        typeCount = 0;
        entries++;
      break;
      
      default:
        printf("Invalid entry during data read in.\n");
    }
  }

  // Close file
  fclose(fp);
  printf("Finished reading.\n");

  // Choose initial centroids
  centroids[0] = area[0];
  centroids[1] = perimeter[0];
  centroids[2] = area[60];
  centroids[3] = perimeter[60];
  centroids[4] = area[120];
  centroids[5] = perimeter[120];

  //TODO Call device function and run for declared amount of epochs
  // Allocate memory to device
  hipMalloc(&a, 180*sizeof(float));
  hipMalloc(&b, 180*sizeof(float));
  hipMalloc(&centroids_d, 6*sizeof(float));
  hipMalloc(&predicted_180_d, 180*sizeof(float));

  // Copy data to device memory
  hipMemcpy(a, area, 180*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b, perimeter, 180*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(centroids_d, centroids, 6*sizeof(float), hipMemcpyHostToDevice);
  
  // Call Device Function - kMeansClustering
  printf("Initiating 1-pass of k-means clustering.\n");
  kMeansClustering<<<1,180>>>(a,b, 180, centroids_d, predicted_180_d);
  printf("Completed.\n");

  // Copy result from device memory
  hipMemcpy(centroids, centroids_d, 6*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(predicted_180, predicted_180_d, 180*sizeof(float), hipMemcpyDeviceToHost);

  checkPredictions(predicted_180, cat, 180);

  // Output prediction and centroids to csv file

  // Predictions
  fp = fopen ("data/output.csv", "w");
  fprintf (fp,"a,b,class\n");
  for(int i = 0; i < 180; i++){
    fprintf (fp," %f,%f,%f\n", area[i], perimeter[i], predicted_180[i]);
  }
  fclose(fp);

  // Actual
  fp = fopen ("data/actual.csv", "w");
  fprintf (fp,"a,b,class\n");
  for(int i = 0; i < 180; i++){
    fprintf (fp,"%f,%f,%f\n", area[i], perimeter[i], cat[i]);
  }
  fclose(fp);

  printf("Initial Centroids:\n");
  for(int i = 0; i < 3; i++){
    printf("(%.4f, %.4f)\n", centroids[2*i], centroids[2*i+1]);
  }

  updateCentroids(area, perimeter, predicted_180, centroids);

  printf("Updated Centroids:\n");
  for(int i = 0; i < 3; i++){
    printf("(%.4f, %.4f)\n", centroids[2*i], centroids[2*i+1]);
  }

  // Output Centroids to csv
  fp = fopen ("data/centroids.csv", "w");
  fprintf (fp,"x,y\n");
  for(int i = 0; i < 3; i++){
    fprintf (fp,"%f,%f\n", centroids[2*i], centroids[2*i+1]);
  }
  fclose(fp);
  

  // Free all memory
  free(area);
  free(perimeter);
  free(compactness);
  free(lenKernel);
  free(widKernel);
  free(asymCoef);
  free(lenKernelGroove);
  free(cat);
  free(t_area);
  free(t_perimeter);
  free(t_compactness);
  free(t_lenKernel);
  free(t_widKernel);
  free(t_asymCoef);
  free(t_lenKernelGroove);
  free(t_cat);
  free(predicted_180);
  free(predicted_30);
  free(centroids);
  hipFree(centroids_d);
  hipFree(a);
  hipFree(b);

  return 0;
}