/******************************************************************************
* FILE: wheat_cluster.cu
* DESCRIPTION:
*   A simple cuda program to compute k-means cluster of a variety of wheat
*   wheat seeds.
* AUTHOR: David Nguyen
* CONTACT: david@knytes.com 
* REVISED: 14/05/2020
******************************************************************************/
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

//TODO Device function for k-means clustering
// Should return the new points of the centroids


int main(){
  // Read in dataset and store into array.
  // 210 total - 70 per class - put 10 random from each class
  float     *area,
            *perimeter,
            *compactness,
            *lenKernel,
            *widKernel,
            *asymCoef,
            *lenKernelGroove,
            *cat;

  // Allocate memory space to variables
  area = (float*)malloc(210*sizeof(float));
  perimeter = (float*)malloc(210*sizeof(float));
  compactness = (float*)malloc(210*sizeof(float));
  lenKernel = (float*)malloc(210*sizeof(float));
  widKernel = (float*)malloc(210*sizeof(float));
  asymCoef = (float*)malloc(210*sizeof(float));
  lenKernelGroove = (float*)malloc(210*sizeof(float));
  cat = (float*)malloc(210*sizeof(float));

  /**
  * Read in data from text file.
  */
  FILE  *fp; // File object
  float fTmp; // temporarily store float
  int   typeCount = 0, // keep count of feature
        entries = 0;
  const char* filename = "seeds_dataset.txt"; // file name

  // Open file
  fp = fopen(filename, "r");

  // Check if file exists
  if (fp == NULL){
    printf("Could not open file %s",filename);
    return 1;
  }

  printf("READING DATASET\n%d: ", entries+1);
  while(entries != 210){
    fscanf(fp,"%6f",&fTmp);
    printf("%.4f ",fTmp);

    switch(typeCount){
      case 0: // area
        area[entries] = fTmp;
        typeCount++;
      break;

      case 1: // perimeter
        perimeter[entries] = fTmp;
        typeCount++;
      break;

      case 2: // compactness
        compactness[entries] = fTmp;
        typeCount++;
      break;

      case 3: // length of kernel
        lenKernel[entries] = fTmp;
        typeCount++;
      break;

      case 4: // width of kernel
        widKernel[entries] = fTmp;
        typeCount++;
      break;

      case 5: // asymmetry coefficient
        asymCoef[entries] = fTmp;
        typeCount++;
      break;

      case 6: // length of kernel groove
        lenKernelGroove[entries] = fTmp;
        typeCount++;
      break;

      case 7: // class
        cat[entries] = fTmp;
        typeCount = 0;
        entries++;
        if(entries != 210){
          printf("\n%d: ", entries+1);
        }else{
          printf("\n");
        }
      break;
      
      default:
        printf("Invalid entry during data read in.\n");
    }
  }

  // Close file
  fclose(fp);

  //TODO Separate into training & testing set

  //TODO Call device function and run for declared amount of epochs

  //TODO Run new centroids against testing set and return accuracy
  

  // Free memory
  // free(area);
  // free(perimeter);
  // free(compactness);
  // free(lenKernel);
  // free(widKernel);
  // free(asymCoef);
  // free(lenKernelGroove);
  // free(class);

  return 0;
}