/******************************************************************************
* FILE: wheat_cluster.cu
* DESCRIPTION:
*   A simple cuda program to compute k-means cluster of a variety of wheat
*   wheat seeds.
* AUTHOR: David Nguyen
* CONTACT: david@knytes.com 
* REVISED: 06/05/2020
******************************************************************************/
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define NUMOFCLASSES  3
#define NUMOFFEATURES 7
#define MAXCHAR 9300


//TODO Device function for k-means clustering
// Should return the new points of the centroids


int main(){
  //TODO Read in dataset and store into array.
  // 210 total - 70 per class - put 10 random from each class
  FILE *fp;
  char str[MAXCHAR];
  char* filename = "seeds_dataset.txt";

  fp = fopen(filename, "r");
  if (fp == NULL){
      printf("Could not open file %s",filename);
      return 1;
  }
  while (fgets(str, MAXCHAR, fp) != NULL)
      printf("%s", str);
  fclose(fp);
  return 0;

  //TODO Separate into training & testing set

  //TODO Call device function and run for declared amount of epochs

  //TODO Run new centroids against testing set and return accuracy
  
}